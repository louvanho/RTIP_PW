#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

#define CV_PI 3.1415926535897932384626433832795

__device__ float Gaussian(int x, int y, float sigma){
    return exp(-(x * x + y * y) / (2 * sigma * sigma)) / (2 * CV_PI * sigma * sigma);
}

__global__ void process(const cv::cuda::PtrStepSz<uchar3> leftImage, const cv::cuda::PtrStepSz<uchar3> rightImage, cv::cuda::PtrStepSz<uchar3> BlurImage, const int kernel_size, const float sigma, int rows, int cols) {

    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dst_x < cols && dst_y < rows) {
        float3 Lsum = {0., 0., 0.};
        float3 Rsum = {0., 0., 0.};
        float counter = 0.;
        for (int h=-kernel_size/2;h<=kernel_size/2;h++){
            for (int w=-kernel_size/2;w<=kernel_size/2;w++){
                if (dst_y+h > 0 && dst_y+h < rows && dst_x+w > 0 && dst_x+w < cols && h*h+w*w <= kernel_size*kernel_size){
                    // Lsum += leftImage.at<cv::Vec3b>(i+h, j+w) * Gaussian(h, w, sigma);
                    // Rsum += rightImage.at<cv::Vec3b>(i+h, j+w) * Gaussian(h, w, sigma);
                    Lsum.x += leftImage(dst_y+h,dst_x+w).x * Gaussian(w, h, sigma);
                    Lsum.y += leftImage(dst_y+h,dst_x+w).y * Gaussian(w, h, sigma);
                    Lsum.z += leftImage(dst_y+h,dst_x+w).z * Gaussian(w, h, sigma);
                    Rsum.x += rightImage(dst_y+h,dst_x+w).x * Gaussian(w, h, sigma);
                    Rsum.y += rightImage(dst_y+h,dst_x+w).y * Gaussian(w, h, sigma);
                    Rsum.z += rightImage(dst_y+h,dst_x+w).z * Gaussian(w, h, sigma);
                    counter += Gaussian(w, h, sigma);
                }
            }
        }

        uchar3 BlurLPixel = {(unsigned char)(Lsum.x / counter), (unsigned char)(Lsum.y / counter), (unsigned char)(Lsum.z / counter)};
        BlurImage(dst_y, dst_x) = BlurLPixel;

        uchar3 BlurRPixel = {(unsigned char)(Rsum.x / counter), (unsigned char)(Rsum.y / counter), (unsigned char)(Rsum.z / counter)};
        BlurImage(dst_y, dst_x + cols) = BlurRPixel;
    }
}

int divUp(int a, int b)
{
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void createGaussianCUDA(cv::cuda::GpuMat& d_Limg, cv::cuda::GpuMat& d_Rimg, cv::cuda::GpuMat& d_result, int kernel_size, float sigma, int block_x, int block_y) {
    const dim3 block(block_x, block_y);
    const dim3 grid(divUp(d_Limg.cols, block.x), divUp(d_Limg.rows, block.y));

    const int rows = d_Limg.rows;
    const int cols = d_Limg.cols;

    process<<<grid, block>>>(d_Limg, d_Rimg, d_result, kernel_size, sigma, rows, cols);
}